
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/SparseToDense.cu"
#else
#include "SparseToDense.h"

extern "C" void scn_DR_(SparseToDense_updateOutput)(
    THLongTensor *inputSize, void **m, THCTensor *input_features,
    THCTensor *output_features, THCITensor *rulesBuffer, long nPlanes) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  long spatialVolume = 1;
  {
    long sz[Dimension + 2];
    sz[0] = _m.grids.begin()->second.size();
    sz[1] = nPlanes; // input_features->size[1];
    for (int i = 0; i < Dimension; i++) {
      auto x = THLongTensor_data(inputSize)[i];
      sz[i + 2] = x;
      spatialVolume *= x;
    }
    THCTensor_(resizeNd)(state, output_features, Dimension + 2, sz, NULL);
    THCTensor_(zero)(state, output_features);
  }
  if (input_features->nDimension == 2) {
    auto _rules = _m.getSparseToDenseRuleBook(inputSize, true);
    uInt nPlanes = input_features->size[1];
    auto iF = THCTensor_(data)(state, input_features);
    auto oF = THCTensor_(data)(state, output_features);
    RULEBOOKITERATOR(
        SparseToDense_ForwardPass<real>(THCState_getCurrentStream(state), iF,
                                        oF, nPlanes, spatialVolume, rbB, nHotB);
        , oF += nPlanes * spatialVolume;)
  }
}
extern "C" void scn_DR_(SparseToDense_updateGradInput)(
    THLongTensor *inputSize, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *d_output_features,
    THCITensor *rulesBuffer) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  if (input_features->nDimension == 2) {
    auto _rules = _m.getSparseToDenseRuleBook(inputSize, true);
    long spatialVolume = THLongTensor_prodall(inputSize);
    uInt nPlanes = d_input_features->size[1];
    auto diF = THCTensor_(data)(state, d_input_features);
    auto doF = THCTensor_(data)(state, d_output_features);
    RULEBOOKITERATOR(SparseToDense_BackwardPass<real>(
                         THCState_getCurrentStream(state), diF, doF, nPlanes,
                         spatialVolume, rbB, nHotB);
                     , doF += nPlanes * spatialVolume;)
  }
}
#endif
